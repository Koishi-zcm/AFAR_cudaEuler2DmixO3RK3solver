#include <stdlib.h>
#include <stdio.h>
#include <ctime>
#include <hip/hip_runtime.h>

#include "GPUevolve.h"
#include "cudaLimiter.h"
#include "cudaRiemannSolver.h"
#include "cudaEulerBCs.h"
#include "cudaFieldOperation.h"


void setDeviceFieldData(
	fieldPointer& devicePtr,
	fieldPointer& hostPtr,
	const int cellsNum,
	const int facesNum,
	const int totalBoundaryFacesNum
)
{
	dim3 blockDim(BLOCKDIM);
	dim3 gridDim((cellsNum + blockDim.x - 1)/blockDim.x);

	hipMalloc((void**)&devicePtr.CELL, sizeof(meshCellData) * (cellsNum + totalBoundaryFacesNum));
	hipMemcpy(devicePtr.CELL, hostPtr.CELL, sizeof(meshCellData) * (cellsNum + totalBoundaryFacesNum), hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.FACE, sizeof(meshFaceData) * (facesNum + totalBoundaryFacesNum));
	hipMemcpy(devicePtr.FACE, hostPtr.FACE, sizeof(meshFaceData) * (facesNum + totalBoundaryFacesNum), hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.cellVolume, sizeof(double) * cellsNum);
	hipMemcpy(devicePtr.cellVolume, hostPtr.cellVolume, sizeof(double) * cellsNum, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.FD, sizeof(basicFieldData) * (cellsNum + totalBoundaryFacesNum));
	hipMemcpy(devicePtr.FD, hostPtr.FD, sizeof(basicFieldData) * (cellsNum + totalBoundaryFacesNum), hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.limiter, sizeof(limiterFieldData) * cellsNum);

	hipMalloc((void**)&devicePtr.shockIndicator, sizeof(int8_t) * 2*(cellsNum + totalBoundaryFacesNum));
	hostPtr.shockIndicator = (int8_t*)malloc(sizeof(int8_t) * 2*(cellsNum + totalBoundaryFacesNum));
	blockDim.x = BLOCKDIM;
	gridDim.x = (2*(cellsNum + totalBoundaryFacesNum) + blockDim.x - 1)/blockDim.x;
	initShockIndicator<<<gridDim, blockDim>>>(devicePtr.shockIndicator, cellsNum, totalBoundaryFacesNum);

	hipMalloc((void**)&devicePtr.cFD, sizeof(conservedFieldData) * cellsNum);
	hipMemcpy(devicePtr.cFD, hostPtr.cFD, sizeof(conservedFieldData) * cellsNum, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.cFDold, sizeof(conservedFieldData) * cellsNum);
	hipMemcpy(devicePtr.cFDold, hostPtr.cFD, sizeof(conservedFieldData) * cellsNum, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.gradFD, sizeof(gradientFieldData) * cellsNum);

	hipMalloc((void**)&devicePtr.Flux, sizeof(basicFluxData) * (facesNum + totalBoundaryFacesNum));

	hipMalloc((void**)&devicePtr.Res, sizeof(residualFieldData) * cellsNum);
	blockDim.x = BLOCKDIM;
	gridDim.x = (cellsNum + blockDim.x - 1)/blockDim.x;
	initResidual<<<gridDim, blockDim>>>(devicePtr.Res, cellsNum);

	hipError_t err = hipGetLastError();
	if(err != hipSuccess) {
		printf("setDeviceFieldData error! %s\n", hipGetErrorString(err));
		std::exit(-1);
	}
}


void setDeviceFieldData(
	fieldPointer& devicePtr,
	fieldPointer& hostPtr,
	const int cellsNum,
	const int facesNum,
	const int patchesNum,
	const int totalBoundaryFacesNum,
	const int maxStencilSize,
	const int maxCompactStencilSize,
	const int maxLocalBlockStencilSize,
	const int maxCompactLocalBlockStencilSize
)
{
	dim3 blockDim(BLOCKDIM);
	dim3 gridDim((cellsNum + blockDim.x - 1)/blockDim.x);

	hipMalloc((void**)&devicePtr.neighbour, sizeof(int) * facesNum*2);
	hipMemcpy(devicePtr.neighbour, hostPtr.neighbour, sizeof(int) * facesNum*2, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.boundaryFacesNum, sizeof(int) * patchesNum);
	hipMemcpy(devicePtr.boundaryFacesNum, hostPtr.boundaryFacesNum, sizeof(int) * patchesNum, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.boundaryFaceNeiLabel, sizeof(int) * totalBoundaryFacesNum);
	hipMemcpy(devicePtr.boundaryFaceNeiLabel, hostPtr.boundaryFaceNeiLabel, sizeof(int) * totalBoundaryFacesNum, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.boundaryFacesType, sizeof(uint8_t) * totalBoundaryFacesNum);
	hipMemcpy(devicePtr.boundaryFacesType, hostPtr.boundaryFacesType, sizeof(uint8_t) * totalBoundaryFacesNum, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.stencilSize, sizeof(uint8_t) * cellsNum);
	hipMemcpy(devicePtr.stencilSize, hostPtr.stencilSize, sizeof(uint8_t) * cellsNum, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.compactStencilSize, sizeof(uint8_t) * cellsNum);
	hipMemcpy(devicePtr.compactStencilSize, hostPtr.compactStencilSize, sizeof(uint8_t) * cellsNum, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.RBFbasis, sizeof(float) * 2*(maxStencilSize+1)*facesNum);
	hipMemcpy(devicePtr.RBFbasis, hostPtr.RBFbasis, sizeof(float) * 2*(maxStencilSize+1)*facesNum, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.faceFD, sizeof(basicFieldData) * 2*facesNum);

	blockDim.x = BLOCKDIM;
	gridDim.x = (cellsNum + blockDim.x - 1)/blockDim.x;

	hipMalloc((void**)&devicePtr.extendStencilSize, sizeof(uint16_t) * gridDim.x);
	hipMemcpy(devicePtr.extendStencilSize, hostPtr.extendStencilSize, sizeof(uint16_t) * gridDim.x, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.compactExtendStencilSize, sizeof(uint16_t) * gridDim.x);
	hipMemcpy(devicePtr.compactExtendStencilSize, hostPtr.compactExtendStencilSize, sizeof(uint16_t) * gridDim.x, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.extendStencil, sizeof(int) * (maxLocalBlockStencilSize - blockDim.x)*gridDim.x);
	hipMemcpy(devicePtr.extendStencil, hostPtr.extendStencil, sizeof(int) * (maxLocalBlockStencilSize - blockDim.x)*gridDim.x, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.localStencil, sizeof(uint16_t) * maxStencilSize*cellsNum);
	hipMemcpy(devicePtr.localStencil, hostPtr.localStencil, sizeof(uint16_t) * maxStencilSize*cellsNum, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.matrix, sizeof(float) * 2*maxCompactStencilSize*cellsNum);

	hipMalloc((void**)&devicePtr.cellFaces, sizeof(int) * maxCompactStencilSize*cellsNum);
	hipMemcpy(devicePtr.cellFaces, hostPtr.cellFaces, sizeof(int) * maxCompactStencilSize*cellsNum, hipMemcpyHostToDevice);

	blockDim.x = 1024;
	gridDim.x = (facesNum + blockDim.x - 1)/blockDim.x;
	hostPtr.minDeltaT = (double*)malloc(sizeof(double)*gridDim.x);
	hipMalloc((void**)&devicePtr.minDeltaT, sizeof(double)*gridDim.x);

	printf("evaluating least square inverse matrix...");
	blockDim.x = BLOCKDIM;
	gridDim.x = (cellsNum + blockDim.x - 1)/blockDim.x;
	const size_t smSize = sizeof(double) * maxCompactLocalBlockStencilSize*2;
	calcLeastSquareMatrix<<<gridDim, blockDim, smSize>>>(
		devicePtr.matrix, devicePtr.localStencil, devicePtr.compactStencilSize,
		devicePtr.CELL, devicePtr.extendStencil, devicePtr.compactExtendStencilSize,
		cellsNum, maxStencilSize, maxCompactStencilSize,
		maxCompactLocalBlockStencilSize, maxLocalBlockStencilSize
	);
	printf("complete!\n");

	hipError_t err = hipGetLastError();
	if(err != hipSuccess) {
		printf("setDeviceFieldData error! %s\n", hipGetErrorString(err));
		std::exit(-1);
	}
}


void freeFieldData(fieldPointer& devicePtr, fieldPointer& hostPtr)
{
	hipFree(devicePtr.CELL);
	hipFree(devicePtr.FACE);
	hipFree(devicePtr.cellVolume);
	hipFree(devicePtr.FD);
	hipFree(devicePtr.cFD);
	hipFree(devicePtr.cFDold);
	hipFree(devicePtr.gradFD);
	hipFree(devicePtr.Flux);
	hipFree(devicePtr.Res);
	hipFree(devicePtr.neighbour);
	hipFree(devicePtr.boundaryFacesNum);
	hipFree(devicePtr.boundaryFaceNeiLabel);
	hipFree(devicePtr.boundaryFacesType);
	hipFree(devicePtr.stencilSize);
	hipFree(devicePtr.extendStencilSize);
	hipFree(devicePtr.compactExtendStencilSize);
	hipFree(devicePtr.extendStencil);
	hipFree(devicePtr.localStencil);
	hipFree(devicePtr.matrix);
	hipFree(devicePtr.cellFaces);
	hipFree(devicePtr.minDeltaT);
}


double adjustTimeStep(
	fieldPointer& devicePtr,
	fieldPointer& hostPtr,
	const int facesNum,
	const double R,
	const double Cv,
	const double CFL
)
{
	dim3 blockDim(1024);
	dim3 gridDim((facesNum + blockDim.x - 1)/blockDim.x);
	setDeltaT<<<gridDim, blockDim>>>(
		devicePtr.minDeltaT, devicePtr.FD,
		devicePtr.CELL, devicePtr.FACE, devicePtr.neighbour,
		facesNum, R, Cv, CFL
	);
	hipDeviceSynchronize();

	hipMemcpy(hostPtr.minDeltaT, devicePtr.minDeltaT, sizeof(double)*gridDim.x, hipMemcpyDeviceToHost);

	double min_deltaT = 1e5;
	for(unsigned i = 0; i < gridDim.x; ++i)
	{
		min_deltaT = min_deltaT > hostPtr.minDeltaT[i] ? hostPtr.minDeltaT[i] : min_deltaT;
	}

	return min_deltaT;
}


void GPUevolve(
	fieldPointer& devicePtr,
	fieldPointer& hostPtr,
	const double R,
	const double Cv,
	const double deltaT,
	const int cellsNum,
	const int facesNum,
	const int totalBoundaryFacesNum,
	const int maxStencilSize,
	const int maxCompactStencilSize,
	const int maxLocalBlockStencilSize,
	const int maxCompactLocalBlockStencilSize
)
{
	const double gamma = R/Cv + 1.0;
	const double beta1[3] = {1.0, 0.75, 0.333333};
	const double beta2[3] = {0.0, 0.25, 0.666667};
	const double beta3[3] = {1.0, 0.25, 0.666667};

	dim3 blockDim(BLOCKDIM);
	dim3 gridDim((cellsNum + blockDim.x - 1)/blockDim.x);

	hipError_t err;

	// three stages SSP Runge-Kutta time evolution
	for(unsigned i = 0; i < 3; ++i)
	{
		blockDim.x = BLOCKDIM;
		gridDim.x = (cellsNum + blockDim.x - 1)/blockDim.x;
		size_t sharedMemSize = sizeof(double)*maxCompactLocalBlockStencilSize*2
			+ sizeof(float)*maxLocalBlockStencilSize*4
			+ sizeof(uint16_t)*blockDim.x*maxStencilSize
			+ sizeof(uint8_t)*blockDim.x*2;
		reconstruct<<<gridDim, blockDim, sharedMemSize>>>(
			devicePtr.faceFD, devicePtr.gradFD, devicePtr.limiter, devicePtr.shockIndicator,
			devicePtr.FD, devicePtr.matrix, devicePtr.RBFbasis, devicePtr.CELL, devicePtr.FACE, devicePtr.cellFaces,
			devicePtr.localStencil, devicePtr.stencilSize, devicePtr.compactStencilSize,
			devicePtr.extendStencil, devicePtr.extendStencilSize, devicePtr.compactExtendStencilSize,
			cellsNum, facesNum, maxStencilSize, maxCompactStencilSize, maxLocalBlockStencilSize, maxCompactLocalBlockStencilSize
		);

		sharedMemSize = sizeof(double)*blockDim.x*4
			+ sizeof(float)*blockDim.x*2
			+ sizeof(uint16_t)*blockDim.x*maxStencilSize
			+ sizeof(uint8_t)*blockDim.x
			+ sizeof(int8_t)*maxLocalBlockStencilSize;
		BVDindicator<<<gridDim, blockDim, sharedMemSize>>>(
			devicePtr.faceFD, devicePtr.gradFD, devicePtr.shockIndicator, devicePtr.limiter,
			devicePtr.FD, devicePtr.CELL, devicePtr.FACE, devicePtr.cellFaces,
			devicePtr.localStencil, devicePtr.stencilSize, devicePtr.compactStencilSize,
			devicePtr.extendStencil, devicePtr.extendStencilSize,
			cellsNum, facesNum, maxStencilSize, maxCompactStencilSize, maxLocalBlockStencilSize
		);

		blockDim.x = BLOCKDIM;
		gridDim.x = (facesNum + blockDim.x - 1)/blockDim.x;
		evaluateFlux<<<gridDim, blockDim>>>(
			devicePtr.Flux, devicePtr.FD, devicePtr.gradFD, devicePtr.limiter,
			devicePtr.faceFD, devicePtr.shockIndicator,
			devicePtr.CELL, devicePtr.FACE, devicePtr.neighbour,
			facesNum, cellsNum, totalBoundaryFacesNum, R, Cv
		);

		gridDim.x = (cellsNum + blockDim.x - 1)/blockDim.x;
		evaluateResidual<<<gridDim, blockDim>>>(
			devicePtr.Res, devicePtr.Flux, devicePtr.cellFaces,
			devicePtr.compactStencilSize,
			cellsNum, maxCompactStencilSize
		);

		gridDim.x = (cellsNum + blockDim.x - 1)/blockDim.x;
		updateFieldData<<<gridDim, blockDim>>>(
			devicePtr.FD, devicePtr.cFD, devicePtr.cFDold,
			devicePtr.Res, devicePtr.cellVolume,
			R, gamma, cellsNum, deltaT,
			beta1[i], beta2[i], beta3[i]
		);

		blockDim.x = BLOCKDIM;
		gridDim.x = (totalBoundaryFacesNum + blockDim.x - 1)/blockDim.x;
		updateBoundaryFieldData<<<gridDim, blockDim>>>(
			devicePtr.FD, devicePtr.CELL, devicePtr.FACE,
			devicePtr.boundaryFaceNeiLabel, devicePtr.boundaryFacesType,
			cellsNum, facesNum, totalBoundaryFacesNum,
			R, Cv
		);
	}

	gridDim.x = (cellsNum + blockDim.x - 1)/blockDim.x;
	storeConservedFieldData<<<gridDim, blockDim>>>(devicePtr.cFDold, devicePtr.cFD, cellsNum);

	hipDeviceSynchronize();

	err = hipGetLastError();
	if(err != hipSuccess) {
		printf("GPUevolve error! %s\n", hipGetErrorString(err));
		std::exit(-1);
	}
}


void copyFieldDataDeviceToHost(
	fieldPointer& hostPtr,
	fieldPointer& devicePtr,
	const int cellsNum,
	const int totalBoundaryFacesNum
)
{
	hipMemcpy(hostPtr.FD, devicePtr.FD, sizeof(basicFieldData)*(cellsNum + totalBoundaryFacesNum), hipMemcpyDeviceToHost);
	hipMemcpy(hostPtr.shockIndicator, devicePtr.shockIndicator, sizeof(int8_t)*2*(cellsNum + totalBoundaryFacesNum), hipMemcpyDeviceToHost);
}