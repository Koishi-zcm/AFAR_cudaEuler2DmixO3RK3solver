#include <stdlib.h>
#include <stdio.h>
#include <ctime>
#include <hip/hip_runtime.h>

#include "GPUevolve.h"
#include "cudaLimiter.h"
#include "cudaRiemannSolver.h"
#include "cudaEulerBCs.h"
#include "cudaFieldOperation.h"


void setDeviceFieldData(
	fieldPointer& devicePtr,
	fieldPointer& hostPtr,
	const int cellsNum,
	const int facesNum,
	const int patchesNum,
	const int totalBoundaryFacesNum,
	const int maxStencilSize,
	const int maxCompactStencilSize,
	const int maxLocalBlockStencilSize
)
{
	dim3 blockDim(BLOCKDIM);
	dim3 gridDim((cellsNum + blockDim.x - 1)/blockDim.x);

	hipMalloc((void**)&devicePtr.CELL, sizeof(meshCellData) * (cellsNum + totalBoundaryFacesNum));
	hipMemcpy(devicePtr.CELL, hostPtr.CELL, sizeof(meshCellData) * (cellsNum + totalBoundaryFacesNum), hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.FACE, sizeof(meshFaceData) * (facesNum + totalBoundaryFacesNum));
	hipMemcpy(devicePtr.FACE, hostPtr.FACE, sizeof(meshFaceData) * (facesNum + totalBoundaryFacesNum), hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.cellVolume, sizeof(double) * cellsNum);
	hipMemcpy(devicePtr.cellVolume, hostPtr.cellVolume, sizeof(double) * cellsNum, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.FD, sizeof(basicFieldData) * (cellsNum + totalBoundaryFacesNum));
	hipMemcpy(devicePtr.FD, hostPtr.FD, sizeof(basicFieldData) * (cellsNum + totalBoundaryFacesNum), hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.limiter, sizeof(limiterFieldData) * cellsNum);

	hipMalloc((void**)&devicePtr.shockIndicator, sizeof(int8_t) * (cellsNum + totalBoundaryFacesNum));
	hipMemcpy(devicePtr.shockIndicator, hostPtr.shockIndicator, sizeof(int8_t) * (cellsNum + totalBoundaryFacesNum), hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.cFD, sizeof(conservedFieldData) * cellsNum);
	hipMemcpy(devicePtr.cFD, hostPtr.cFD, sizeof(conservedFieldData) * cellsNum, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.cFDold, sizeof(conservedFieldData) * cellsNum);
	hipMemcpy(devicePtr.cFDold, hostPtr.cFD, sizeof(conservedFieldData) * cellsNum, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.gradFD, sizeof(gradientFieldData) * cellsNum);

	hipMalloc((void**)&devicePtr.Flux, sizeof(basicFluxData) * (facesNum + totalBoundaryFacesNum));
	hipMemcpy(devicePtr.Flux, hostPtr.Flux, sizeof(basicFluxData) * (facesNum + totalBoundaryFacesNum), hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.Res, sizeof(residualFieldData) * cellsNum);
	hipMemcpy(devicePtr.Res, devicePtr.Res, sizeof(residualFieldData) * cellsNum, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.neighbour, sizeof(int) * facesNum*2);
	hipMemcpy(devicePtr.neighbour, hostPtr.neighbour, sizeof(int) * facesNum*2, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.boundaryFacesNum, sizeof(int) * patchesNum);
	hipMemcpy(devicePtr.boundaryFacesNum, hostPtr.boundaryFacesNum, sizeof(int) * patchesNum, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.boundaryFaceNeiLabel, sizeof(int) * totalBoundaryFacesNum);
	hipMemcpy(devicePtr.boundaryFaceNeiLabel, hostPtr.boundaryFaceNeiLabel, sizeof(int) * totalBoundaryFacesNum, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.boundaryFacesType, sizeof(uint8_t) * totalBoundaryFacesNum);
	hipMemcpy(devicePtr.boundaryFacesType, hostPtr.boundaryFacesType, sizeof(uint8_t) * totalBoundaryFacesNum, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.stencilSize, sizeof(uint8_t) * cellsNum);
	hipMemcpy(devicePtr.stencilSize, hostPtr.stencilSize, sizeof(uint8_t) * cellsNum, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.compactStencilSize, sizeof(uint8_t) * cellsNum);
	hipMemcpy(devicePtr.compactStencilSize, hostPtr.compactStencilSize, sizeof(uint8_t) * cellsNum, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.RBFbasis, sizeof(float) * 2*(maxStencilSize+1)*facesNum);
	hipMemcpy(devicePtr.RBFbasis, hostPtr.RBFbasis, sizeof(float) * 2*(maxStencilSize+1)*facesNum, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.faceFD, sizeof(basicFieldData) * 2*facesNum);

	blockDim.x = BLOCKDIM;
	gridDim.x = (cellsNum + blockDim.x - 1)/blockDim.x;

	hipMalloc((void**)&devicePtr.extendStencilSize, sizeof(uint16_t) * gridDim.x);
	hipMemcpy(devicePtr.extendStencilSize, hostPtr.extendStencilSize, sizeof(uint16_t) * gridDim.x, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.extendStencil, sizeof(int) * (maxLocalBlockStencilSize - blockDim.x)*gridDim.x);
	hipMemcpy(devicePtr.extendStencil, hostPtr.extendStencil, sizeof(int) * (maxLocalBlockStencilSize - blockDim.x)*gridDim.x, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.localStencil, sizeof(uint16_t) * maxStencilSize*cellsNum);
	hipMemcpy(devicePtr.localStencil, hostPtr.localStencil, sizeof(uint16_t) * maxStencilSize*cellsNum, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.matrix, sizeof(float) * 2*maxCompactStencilSize*cellsNum);

	hipMalloc((void**)&devicePtr.cellFaces, sizeof(int) * maxCompactStencilSize*cellsNum);
	hipMemcpy(devicePtr.cellFaces, hostPtr.cellFaces, sizeof(int) * maxCompactStencilSize*cellsNum, hipMemcpyHostToDevice);

	hipMalloc((void**)&devicePtr.faceDirection, sizeof(int8_t) * maxCompactStencilSize*cellsNum);
	hipMemcpy(devicePtr.faceDirection, hostPtr.faceDirection, sizeof(int8_t) * maxCompactStencilSize*cellsNum, hipMemcpyHostToDevice);

	blockDim.x = 1024;
	gridDim.x = (facesNum + blockDim.x - 1)/blockDim.x;
	hostPtr.minDeltaT = (double*)malloc(sizeof(double)*gridDim.x);
	hipMalloc((void**)&devicePtr.minDeltaT, sizeof(double)*gridDim.x);

	printf("evaluating least square inverse matrix...");
	blockDim.x = BLOCKDIM;
	gridDim.x = (cellsNum + blockDim.x - 1)/blockDim.x;
	const size_t smSize = sizeof(double) * maxLocalBlockStencilSize*2;
	calcLeastSquareMatrix<<<gridDim, blockDim, smSize>>>(
		devicePtr.matrix, devicePtr.localStencil, devicePtr.compactStencilSize,
		devicePtr.CELL, devicePtr.extendStencil, devicePtr.extendStencilSize,
		cellsNum, maxStencilSize, maxCompactStencilSize, maxLocalBlockStencilSize
	);
	printf("complete!\n");

	hipError_t err = hipGetLastError();
	if(err != hipSuccess) {
		printf("setDeviceFieldData error! %s\n", hipGetErrorString(err));
		std::exit(-1);
	}
}


void freeFieldData(fieldPointer& devicePtr, fieldPointer& hostPtr)
{
	hipFree(devicePtr.CELL);
	hipFree(devicePtr.FACE);
	hipFree(devicePtr.cellVolume);
	hipFree(devicePtr.FD);
	hipFree(devicePtr.cFD);
	hipFree(devicePtr.cFDold);
	hipFree(devicePtr.gradFD);
	hipFree(devicePtr.Flux);
	hipFree(devicePtr.Res);
	hipFree(devicePtr.neighbour);
	hipFree(devicePtr.boundaryFacesNum);
	hipFree(devicePtr.boundaryFaceNeiLabel);
	hipFree(devicePtr.boundaryFacesType);
	hipFree(devicePtr.stencilSize);
	hipFree(devicePtr.extendStencilSize);
	hipFree(devicePtr.extendStencil);
	hipFree(devicePtr.localStencil);
	hipFree(devicePtr.matrix);
	hipFree(devicePtr.cellFaces);
	hipFree(devicePtr.minDeltaT);
}


double adjustTimeStep(
	fieldPointer& devicePtr,
	fieldPointer& hostPtr,
	const int facesNum,
	const double R,
	const double Cv,
	const double CFL
)
{
	dim3 blockDim(1024);
	dim3 gridDim((facesNum + blockDim.x - 1)/blockDim.x);
	setDeltaT<<<gridDim, blockDim>>>(
		devicePtr.minDeltaT, devicePtr.FD,
		devicePtr.CELL, devicePtr.FACE, devicePtr.neighbour,
		facesNum, R, Cv, CFL
	);
	hipDeviceSynchronize();

	hipMemcpy(hostPtr.minDeltaT, devicePtr.minDeltaT, sizeof(double)*gridDim.x, hipMemcpyDeviceToHost);

	double min_deltaT = 1e5;
	for(unsigned i = 0; i < gridDim.x; ++i)
	{
		min_deltaT = min_deltaT > hostPtr.minDeltaT[i] ? hostPtr.minDeltaT[i] : min_deltaT;
	}

	return min_deltaT;
}


void GPUevolve(
	fieldPointer& devicePtr,
	fieldPointer& hostPtr,
	const double R,
	const double Cv,
	const double deltaT,
	const int cellsNum,
	const int facesNum,
	const int totalBoundaryFacesNum,
	const int maxStencilSize,
	const int maxCompactStencilSize,
	const int maxLocalBlockStencilSize
)
{
	const double gamma = R/Cv + 1.0;
	const double beta1[3] = {1.0, 0.75, 0.333333};
	const double beta2[3] = {0.0, 0.25, 0.666667};
	const double beta3[3] = {1.0, 0.25, 0.666667};

	dim3 blockDim(BLOCKDIM);
	dim3 gridDim((cellsNum + blockDim.x - 1)/blockDim.x);

	hipError_t err;

	// three stages SSP Runge-Kutta time evolution
	for(unsigned i = 0; i < 3; ++i)
	{
		blockDim.x = BLOCKDIM;
		gridDim.x = (cellsNum + blockDim.x - 1)/blockDim.x;
		size_t sharedMemSize = sizeof(double)*maxLocalBlockStencilSize*2
			+ sizeof(float)*maxLocalBlockStencilSize*4
			+ sizeof(int)*maxLocalBlockStencilSize
			+ sizeof(uint16_t)*blockDim.x*maxStencilSize
			+ sizeof(uint8_t)*blockDim.x*2;
		reconstruct<<<gridDim, blockDim, sharedMemSize>>>(
			devicePtr.faceFD, devicePtr.gradFD, devicePtr.limiter, devicePtr.shockIndicator,
			devicePtr.FD, devicePtr.matrix, devicePtr.RBFbasis, devicePtr.CELL, devicePtr.FACE, devicePtr.cellFaces,
			devicePtr.localStencil, devicePtr.stencilSize, devicePtr.compactStencilSize,
			devicePtr.extendStencil, devicePtr.extendStencilSize,
			cellsNum, maxStencilSize, maxCompactStencilSize, maxLocalBlockStencilSize
		);

		sharedMemSize = sizeof(double)*blockDim.x*4
			+ sizeof(float)*blockDim.x*2
			+ sizeof(int)*maxLocalBlockStencilSize
			+ sizeof(uint16_t)*blockDim.x*maxStencilSize
			+ sizeof(uint8_t)*blockDim.x
			+ sizeof(int8_t)*maxLocalBlockStencilSize;
		BVDindicator<<<gridDim, blockDim, sharedMemSize>>>(
			devicePtr.faceFD, devicePtr.gradFD, devicePtr.shockIndicator, devicePtr.limiter,
			devicePtr.FD, devicePtr.CELL, devicePtr.FACE, devicePtr.cellFaces,
			devicePtr.localStencil, devicePtr.stencilSize, devicePtr.compactStencilSize,
			devicePtr.extendStencil, devicePtr.extendStencilSize,
			cellsNum, maxStencilSize, maxCompactStencilSize, maxLocalBlockStencilSize
		);

		blockDim.x = BLOCKDIM;
		gridDim.x = (facesNum + blockDim.x - 1)/blockDim.x;
		evaluateFlux<<<gridDim, blockDim>>>(
			devicePtr.Flux, devicePtr.FD, devicePtr.gradFD, devicePtr.limiter,
			devicePtr.faceFD, devicePtr.shockIndicator,
			devicePtr.CELL, devicePtr.FACE, devicePtr.neighbour,
			facesNum, cellsNum, totalBoundaryFacesNum, R, Cv
		);

		gridDim.x = (cellsNum + blockDim.x - 1)/blockDim.x;
		evaluateResidual<<<gridDim, blockDim>>>(
			devicePtr.Res, devicePtr.Flux, devicePtr.cellFaces,
			devicePtr.compactStencilSize, devicePtr.faceDirection,
			cellsNum, maxCompactStencilSize
		);

		gridDim.x = (cellsNum + blockDim.x - 1)/blockDim.x;
		updateFieldData<<<gridDim, blockDim>>>(
			devicePtr.FD, devicePtr.cFD, devicePtr.cFDold,
			devicePtr.Res, devicePtr.cellVolume,
			R, gamma, cellsNum, deltaT,
			beta1[i], beta2[i], beta3[i]
		);

		blockDim.x = BLOCKDIM;
		gridDim.x = (totalBoundaryFacesNum + blockDim.x - 1)/blockDim.x;
		updateBoundaryFieldData<<<gridDim, blockDim>>>(
			devicePtr.FD, devicePtr.CELL, devicePtr.FACE,
			devicePtr.boundaryFaceNeiLabel, devicePtr.boundaryFacesType,
			cellsNum, facesNum, totalBoundaryFacesNum,
			R, Cv
		);
	}

	gridDim.x = (cellsNum + blockDim.x - 1)/blockDim.x;
	storeConservedFieldData<<<gridDim, blockDim>>>(devicePtr.cFDold, devicePtr.cFD, cellsNum);

	hipDeviceSynchronize();

	err = hipGetLastError();
	if(err != hipSuccess) {
		printf("GPUevolve error! %s\n", hipGetErrorString(err));
		std::exit(-1);
	}
}


void copyFieldDataDeviceToHost(
	fieldPointer& hostPtr,
	fieldPointer& devicePtr,
	const int cellsNum,
	const int totalBoundaryFacesNum
)
{
	hipMemcpy(hostPtr.FD, devicePtr.FD, sizeof(basicFieldData)*(cellsNum + totalBoundaryFacesNum), hipMemcpyDeviceToHost);
	hipMemcpy(hostPtr.shockIndicator, devicePtr.shockIndicator, sizeof(int8_t)*(cellsNum + totalBoundaryFacesNum), hipMemcpyDeviceToHost);
}